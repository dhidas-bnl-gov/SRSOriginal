#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////
//
// Dean Andrew Hidas <dhidas@bnl.gov>
//
// Created on: Tue May 24 11:33:19 EDT 2016
//
////////////////////////////////////////////////////////////////////

#include "SRS_Cuda.h"

#include "SRS.h"

#include <cmath>
#include <complex>
#include <fstream>
#include <sstream>

#include "TVector3DC.h"
#include "TBField1DZRegularized.h"
#include "TBField3DZRegularized.h"
#include "TSpectrumContainer.h"






__global__ void Orthogonal(double *a, double *b)
{
  // Return a vector which is orthogonal vector a
  double xx = a[0] < 0.0 ? -a[0] : a[0];
  double yy = a[1] < 0.0 ? -a[1] : a[1];
  double zz = a[2] < 0.0 ? -a[2] : a[2];
  if (xx < yy) {
    if (xx < zz) {
      b[0] = 0;
      b[1] = a[2];
      b[2] = -a[1];
    } else {
      b[0] = a[1];
      b[1] = -a[0];
      b[2] = 0;
    }
  } else {
    if (yy < zz) {
      b[0] = -a[2];
      b[1] = 0;
      b[2] = a[0];
    } else {
      b[0] = a[1];
      b[1] = -a[0];
      b[2] = 0;
    }
  }
}



void PowerDensityGPU_Single (double *x, double *y, double *z, double *bx, double *by, double *bz, double *aocx, double *aocy, double *aocz, double *sx, double *sy, double *sz, double *snx, double *sny, double *snz, double *dt, int *nt, int *ns, double *power_density)
{
  for (int is = 0; is < *ns; ++is) {



  // Observation point
  double const OX = sx[is];
  double const OY = sy[is];
  double const OZ = sz[is];

  // Normal vector from input
  double const NormalX = snx[is];
  double const NormalY = sny[is];
  double const NormalZ = snz[is];

  double Sum = 0;

  for (int i = 0; i < *nt; ++i) {

    // Normal vector in direction of observation point
    double const R1 = sqrt( pow(OX - x[i], 2) + pow(OY - y[i], 2) + pow(OZ - z[i], 2) );
    double const N1X = (OX - x[i]) / R1;
    double const N1Y = (OY - y[i]) / R1;
    double const N1Z = (OZ - z[i]) / R1;

    // Surface normal dot with vector normal
    double const N1DotNormal = N1X * NormalX + N1Y * NormalY + N1Z * NormalZ;

    // Orthogonal vector 2 & 3
    double N2X;
    double N2Y;
    double N2Z;

    double const xx = N1X < 0.0 ? -N1X : N1X;
    double const yy = N1Y < 0.0 ? -N1Y : N1Y;
    double const zz = N1Z < 0.0 ? -N1Z : N1Z;
    if (xx < yy) {
      if (xx < zz) {
        N2X = 0;
        N2Y = N1Z;
        N2Z = -N1Y;
      } else {
        N2X = N1Y;
        N2Y = -N1X;
        N2Z = 0;
      }
    } else {
      if (yy < zz) {
        N2X = -N1Z;
        N2Y = 0;
        N2Z = N1X;
      } else {
        N2X = N1Y;
        N2Y = -N1X;
        N2Z = 0;
      }
    }
    double const R2 = sqrt(N2X * N2X + N2Y * N2Y + N2Z * N2Z);
    N2X /= R2;
    N2Y /= R2;
    N2Z /= R2;

    // Ortohgonal vector N3
    double const N3X = N1Y * N2Z - N1Z * N2Y;
    double const N3Y = N1Z * N2X - N1X * N2Z;
    double const N3Z = N1X * N2Y - N1Y * N2X;





    double const x1 = N1X - bx[i];
    double const y1 = N1Y - by[i];
    double const z1 = N1Z - bz[i];

    double const x2 = y1 * aocz[i] - z1 * aocy[i];
    double const y2 = z1 * aocx[i] - x1 * aocz[i];
    double const z2 = x1 * aocy[i] - y1 * aocx[i];

    // Numerator = N1.Cross( ( (N1 - B).Cross((AoverC)) ) );
    double const x3 = N1Y * z2 - N1Z * y2;
    double const y3 = N1Z * x2 - N1X * z2;
    double const z3 = N1X * y2 - N1Y * x2;

    double const BdotN1 = bx[i] * N1X + by[i] * N1Y + bz[i] * N1Z;
    double const Denominator = pow(1. - BdotN1, 5);

    Sum += pow( x3 * N2X + y3 * N2Y + z3 * N2Z, 2) / Denominator / (R1 * R1) * N1DotNormal;
    Sum += pow( x3 * N3X + y3 * N3Y + z3 * N3Z, 2) / Denominator / (R1 * R1) * N1DotNormal;
  }

  power_density[is] = Sum * (*dt);
  }

  return;
}

__global__ void SRS_Cuda_PowerDensityGPU (double *x, double *y, double *z, double *bx, double *by, double *bz, double *aocx, double *aocy, double *aocz, double *sx, double *sy, double *sz, double *snx, double *sny, double *snz, double *dt, int *nt, int *ns, double *power_density)
{
  // Get surface id from block and thread number
  int is = threadIdx.x + blockIdx.x * blockDim.x;

  if (is >= *ns) {
    return;
  }


  for (int i = 0; i < *nt; ++i) {
    power_density[is] = is;

  }



  // If you could copy int ultra-fast memory, something like this:
  //__shared__ double temp[6144];
  //if (threadIdx.x == 0) {
  //  for (int i = 0; i < *nt; ++i) {
  //    if (i <= 6144) {
  //      break;
  //    }
  //    temp[i] = x[i];
  //  }
  //}
  // __syncthreads();



  // Observation point
  double const OX = sx[is];
  double const OY = sy[is];
  double const OZ = sz[is];

  // Normal vector from input
  double const NormalX = snx[is];
  double const NormalY = sny[is];
  double const NormalZ = snz[is];

  double Sum = 0;

  for (int i = 0; i < *nt; ++i) {

    // Normal vector in direction of observation point
    double const R1 = sqrt( pow(OX - x[i], 2) + pow(OY - y[i], 2) + pow(OZ - z[i], 2) );
    double const N1X = (OX - x[i]) / R1;
    double const N1Y = (OY - y[i]) / R1;
    double const N1Z = (OZ - z[i]) / R1;

    // Surface normal dot with vector normal
    double const N1DotNormal = N1X * NormalX + N1Y * NormalY + N1Z * NormalZ;

    // Orthogonal vector 2 & 3
    double N2X;
    double N2Y;
    double N2Z;

    double const xx = N1X < 0.0 ? -N1X : N1X;
    double const yy = N1Y < 0.0 ? -N1Y : N1Y;
    double const zz = N1Z < 0.0 ? -N1Z : N1Z;
    if (xx < yy) {
      if (xx < zz) {
        N2X = 0;
        N2Y = N1Z;
        N2Z = -N1Y;
      } else {
        N2X = N1Y;
        N2Y = -N1X;
        N2Z = 0;
      }
    } else {
      if (yy < zz) {
        N2X = -N1Z;
        N2Y = 0;
        N2Z = N1X;
      } else {
        N2X = N1Y;
        N2Y = -N1X;
        N2Z = 0;
      }
    }
    double const R2 = sqrt(N2X * N2X + N2Y * N2Y + N2Z * N2Z);
    N2X /= R2;
    N2Y /= R2;
    N2Z /= R2;

    // Ortohgonal vector N3
    double const N3X = N1Y * N2Z - N1Z * N2Y;
    double const N3Y = N1Z * N2X - N1X * N2Z;
    double const N3Z = N1X * N2Y - N1Y * N2X;





    double const x1 = N1X - bx[i];
    double const y1 = N1Y - by[i];
    double const z1 = N1Z - bz[i];

    double const x2 = y1 * aocz[i] - z1 * aocy[i];
    double const y2 = z1 * aocx[i] - x1 * aocz[i];
    double const z2 = x1 * aocy[i] - y1 * aocx[i];

    // Numerator = N1.Cross( ( (N1 - B).Cross((AoverC)) ) );
    double const x3 = N1Y * z2 - N1Z * y2;
    double const y3 = N1Z * x2 - N1X * z2;
    double const z3 = N1X * y2 - N1Y * x2;

    double const BdotN1 = bx[i] * N1X + by[i] * N1Y + bz[i] * N1Z;
    double const Denominator = pow(1. - BdotN1, 5);

    Sum += pow( x3 * N2X + y3 * N2Y + z3 * N2Z, 2) / Denominator / (R1 * R1) * N1DotNormal;
    Sum += pow( x3 * N3X + y3 * N3Y + z3 * N3Z, 2) / Denominator / (R1 * R1) * N1DotNormal;
  }

  power_density[is] = Sum * (*dt);

  return;
}



void SRS_Cuda_CalculatePowerDensityGPU (TParticleA& Particle, TSurfacePoints const& Surface, T3DScalarContainer& PowerDensityContainer, int const Dimension, bool const Directional, std::string const& OutFileName)
{

  int ngpu = 0;
  hipGetDeviceCount(&ngpu);
  if (ngpu == 0) {
    throw std::invalid_argument("No GPU found");
  }

  std::cout << "ngpu " << ngpu << std::endl;




  // Grab the Trajectory
  TParticleTrajectoryPoints& T = Particle.GetTrajectory();

  // Number of points in Trajectory
  size_t const NTPoints = T.GetNPoints();

  // Timestep from trajectory
  double const DeltaT = T.GetDeltaT();

  double *x     = new double[NTPoints];
  double *y     = new double[NTPoints];
  double *z     = new double[NTPoints];
  double *bx    = new double[NTPoints];
  double *by    = new double[NTPoints];
  double *bz    = new double[NTPoints];
  double *aocx  = new double[NTPoints];
  double *aocy  = new double[NTPoints];
  double *aocz  = new double[NTPoints];

  size_t const NSPoints = PowerDensityContainer.GetNPoints();

  double *sx     = new double[NSPoints];
  double *sy     = new double[NSPoints];
  double *sz     = new double[NSPoints];

  double *snx    = new double[NSPoints];
  double *sny    = new double[NSPoints];
  double *snz    = new double[NSPoints];

  double *power_density = new double[NSPoints];


  for (size_t i = 0; i < NTPoints; ++i) {
    x[i] = T.GetX(i).GetX();
    y[i] = T.GetX(i).GetY();
    z[i] = T.GetX(i).GetZ();

    bx[i] = T.GetB(i).GetX();
    by[i] = T.GetB(i).GetY();
    bz[i] = T.GetB(i).GetZ();

    aocx[i] = T.GetAoverC(i).GetX();
    aocy[i] = T.GetAoverC(i).GetY();
    aocz[i] = T.GetAoverC(i).GetZ();
  }



  for (size_t i = 0; i < NSPoints; ++i) {
    sx[i] = Surface.GetPoint(i).GetX();
    sy[i] = Surface.GetPoint(i).GetY();
    sz[i] = Surface.GetPoint(i).GetZ();

    snx[i] = Surface.GetPoint(i).GetNormalX();
    sny[i] = Surface.GetPoint(i).GetNormalY();
    snz[i] = Surface.GetPoint(i).GetNormalZ();
  }



  double *d_x, *d_y, *d_z;
  double *d_bx, *d_by, *d_bz;
  double *d_aocx, *d_aocy, *d_aocz;
  double *d_sx, *d_sy, *d_sz;
  double *d_snx, *d_sny, *d_snz;
  double *d_power_density;
  double *d_dt;
  int    *d_nt, *d_ns;

  int const size_x = NTPoints * sizeof(double);
  int const size_s = NSPoints * sizeof(double);

  hipMalloc((void **) &d_x, size_x);
  hipMalloc((void **) &d_y, size_x);
  hipMalloc((void **) &d_z, size_x);

  hipMalloc((void **) &d_bx, size_x);
  hipMalloc((void **) &d_by, size_x);
  hipMalloc((void **) &d_bz, size_x);

  hipMalloc((void **) &d_aocx, size_x);
  hipMalloc((void **) &d_aocy, size_x);
  hipMalloc((void **) &d_aocz, size_x);

  hipMalloc((void **) &d_sx, size_s);
  hipMalloc((void **) &d_sy, size_s);
  hipMalloc((void **) &d_sz, size_s);

  hipMalloc((void **) &d_snx, size_s);
  hipMalloc((void **) &d_sny, size_s);
  hipMalloc((void **) &d_snz, size_s);

  hipMalloc((void **) &d_power_density, size_s);

  hipMalloc((void **) &d_dt, sizeof(double));

  hipMalloc((void **) &d_nt, sizeof(int));
  hipMalloc((void **) &d_ns, sizeof(int));


  hipMemcpy(d_x, &x, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_y, &y, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_z, &z, size_x, hipMemcpyHostToDevice);

  hipMemcpy(d_bx, &bx, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_by, &by, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_bz, &bz, size_x, hipMemcpyHostToDevice);

  hipMemcpy(d_aocx, &aocx, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_aocy, &aocy, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_aocz, &aocz, size_x, hipMemcpyHostToDevice);

  hipMemcpy(d_sx, &sx, size_s, hipMemcpyHostToDevice);
  hipMemcpy(d_sy, &sy, size_s, hipMemcpyHostToDevice);
  hipMemcpy(d_sz, &sz, size_s, hipMemcpyHostToDevice);

  hipMemcpy(d_snx, &snx, size_s, hipMemcpyHostToDevice);
  hipMemcpy(d_sny, &sny, size_s, hipMemcpyHostToDevice);
  hipMemcpy(d_snz, &snz, size_s, hipMemcpyHostToDevice);

  hipMemcpy(d_dt, &DeltaT, sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_nt, &NTPoints, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_ns, &NSPoints, sizeof(int), hipMemcpyHostToDevice);


  // Send computation to gpu
  int const NThreadsPerBlock = 512;
  int const NBlocks = NSPoints / NThreadsPerBlock + 1;
  SRS_Cuda_PowerDensityGPU<<<NBlocks, NThreadsPerBlock>>>(d_x, d_y, d_z, d_bx, d_by, d_bz, d_aocx, d_aocy, d_aocz, d_sx, d_sy, d_sz, d_snx, d_sny, d_snz, d_dt, d_nt, d_ns, d_power_density);

  // Copy result back from GPU
  hipMemcpy(&power_density, d_power_density, size_s, hipMemcpyDeviceToHost);



  // Add result to power density container
  for (size_t i = 0; i < NSPoints; ++i) {
    PowerDensityContainer.AddPoint(TVector3D(sx[i], sy[i], sz[i]), power_density[i] * fabs(Particle.GetQ() * Particle.GetCurrent()) / (16 * TSRS::Pi2() * TSRS::Epsilon0() * TSRS::C()));
  }


  // Free all gpu memory
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);

  hipFree(d_bx);
  hipFree(d_by);
  hipFree(d_bz);

  hipFree(d_aocx);
  hipFree(d_aocy);
  hipFree(d_aocz);

  hipFree(d_sx);
  hipFree(d_sy);
  hipFree(d_sz);

  hipFree(d_snx);
  hipFree(d_sny);
  hipFree(d_snz);

  hipFree(d_dt);
  hipFree(d_nt);
  hipFree(d_ns);

  hipFree(d_power_density);





  // Free all heap memory
  delete [] x;
  delete [] y;
  delete [] z;

  delete [] bx;
  delete [] by;
  delete [] bz;

  delete [] aocx;
  delete [] aocy;
  delete [] aocz;

  delete [] sx;
  delete [] sy;
  delete [] sz;

  delete [] snx;
  delete [] sny;
  delete [] snz;

  delete [] power_density;

  return;
}





